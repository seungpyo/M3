
#include <hip/hip_runtime.h>
// Device code
extern "C" __global__ void m3shell_memset_kernel(char *ptr, int sz, char val)
{
    // Dummy kernel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (; idx < sz; idx += (gridDim.x * blockDim.x)) {
        ptr[idx] = val;
    }
}
